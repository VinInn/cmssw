#include "hip/hip_runtime.h"
#include "HeterogeneousCore/CUDAUtilities/interface/prefixScan.h"
#include <cassert>

template<typename T>
__global__
void testPrefixScan(uint32_t size) {

  __shared__ T ws[32];
  __shared__ T c[1024];
  auto first = threadIdx.x;
  for (auto i=first; i<size; i+=blockDim.x) c[i]=1;

  blockPrefixScan(c, size, ws);

  assert(1==c[0]);
  for (auto i=first; i<size-1; i+=blockDim.x) {
    if (c[i+1]!=c[i]+1) printf("failed %d %d %d: %d %d\n",size, i, blockDim.x, c[i],c[i+1]);
    assert(c[i+1]==c[i]+1); assert(c[i]==i+1);
  }
}

#include <hipcub/hipcub.hpp> 


__global__
void  init(uint32_t  * v, uint32_t  val, uint32_t n) {
    auto i  = blockIdx.x * blockDim.x + threadIdx.x;
    if(i<n) v[i]=val;
    if (i==0) printf("init\n");
}

__global__
void  verify(uint32_t  const * v, uint32_t n) {
    auto i  = blockIdx.x * blockDim.x + threadIdx.x;
    if(i<n) assert(v[i]==i+1);
    if (i==0) printf("verify\n");
}


int main() {
  
  for(int bs=32; bs<=1024; bs+=32)
  for (int j=1;j<=1024; ++j) {
   testPrefixScan<uint16_t><<<1,bs>>>(j);
   testPrefixScan<float><<<1,bs>>>(j);
  }
  hipDeviceSynchronize();

  // Declare, allocate, and initialize device-accessible pointers for input and output
  int  num_items = 10000;
  uint32_t  *d_in;         
  uint32_t  *d_out;


  hipMalloc(&d_in,num_items*sizeof(uint32_t));
  // hipMalloc(&d_out,num_items*sizeof(uint32_t));

  d_out = d_in;
 
  auto nthreads = 256;
  auto nblocks = (num_items + nthreads - 1) / nthreads;

  init<<<nblocks, nthreads, 0>>>(d_in, 1, num_items);

  // Determine temporary device storage requirements for inclusive prefix sum
  void     *d_temp_storage = nullptr;
  size_t   temp_storage_bytes = 0;
  hipcub::DeviceScan::InclusiveSum(d_temp_storage, temp_storage_bytes, d_in, d_out, num_items);

  std::cout << "temp storage " << temp_storage_bytes << std::endl;

  // Allocate temporary storage for inclusive prefix sum
  // fake larger ws already available
  temp_storage_bytes *=8;
  hipMalloc(&d_temp_storage, temp_storage_bytes);
  std::cout << "temp storage " << temp_storage_bytes << std::endl;
  // Run inclusive prefix sum
  CubDebugExit(hipcub::DeviceScan::InclusiveSum(d_temp_storage, temp_storage_bytes, d_in, d_out, num_items));
  std::cout << "temp storage " << temp_storage_bytes << std::endl;

  verify<<<nblocks, nthreads, 0>>>(d_out, num_items);

  hipDeviceSynchronize();

  return 0;
}
