#include "hip/hip_runtime.h"
#define GPU_DEBUG
#include "RecoPixelVertexing/PixelTriplets/plugins/CAHitNtupletGeneratorKernelsImpl.h"

template <>
void CAHitNtupletGeneratorKernelsGPU::fillHitDetIndices(HitsView const *hv, TkSoA *tracks_d, hipStream_t cudaStream) {
  auto blockSize = 128;
  auto numberOfBlocks = (HitContainer::capacity() + blockSize - 1) / blockSize;

  kernel_fillHitDetIndices<<<numberOfBlocks, blockSize, 0, cudaStream>>>(
      &tracks_d->hitIndices, hv, &tracks_d->detIndices);
  cudaCheck(hipGetLastError());
#ifdef GPU_DEBUG
  hipDeviceSynchronize();
  cudaCheck(hipGetLastError());
#endif
}

template <>
void CAHitNtupletGeneratorKernelsGPU::launchKernels(HitsOnCPU const &hh, TkSoA *tracks_d, hipStream_t cudaStream) {
  // these are pointer on GPU!
  auto *tuples_d = &tracks_d->hitIndices;
  auto *quality_d = (Quality *)(&tracks_d->m_quality);

  // zero tuples
  cms::cuda::launchZero(tuples_d, cudaStream);

  auto nhits = hh.nHits();
  assert(nhits <= pixelGPUConstants::maxNumberOfHits);

  // std::cout << "N hits " << nhits << std::endl;
  // if (nhits<2) std::cout << "too few hits " << nhits << std::endl;

  //
  // applying conbinatoric cleaning such as fishbone at this stage is too expensive
  //

  auto nthTot = 64;
  auto stride = 4;
  int blockSize = nthTot / stride;
  int numberOfBlocks = (3 * m_params.maxNumberOfDoublets_ / 4 + blockSize - 1) / blockSize;
  auto rescale = numberOfBlocks / 65536;
  blockSize *= (rescale + 1);
  numberOfBlocks = (3 * m_params.maxNumberOfDoublets_ / 4 + blockSize - 1) / blockSize;
  assert(numberOfBlocks < 65536);
  assert(blockSize > 0 && 0 == blockSize % 16);
  dim3 blks(1, numberOfBlocks, 1);
  dim3 thrs(stride, blockSize, 1);

  kernel_connect<<<blks, thrs, 0, cudaStream>>>(
      device_hitTuple_apc_,
      device_hitToTuple_apc_,  // needed only to be reset, ready for next kernel
      hh.view(),
      device_theCells_.get(),
      device_nCells_,
      device_theCellNeighbors_,
      device_isOuterHitOfCell_.get(),
      m_params.hardCurvCut_,
      m_params.ptmin_,
      m_params.CAThetaCutBarrel_,
      m_params.CAThetaCutForward_,
      m_params.dcaCutInnerTriplet_,
      m_params.dcaCutOuterTriplet_);
  cudaCheck(hipGetLastError());

  if (nhits > 1 && m_params.earlyFishbone_) {
    auto nthTot = 128;
    auto stride = 16;
    auto blockSize = nthTot / stride;
    auto numberOfBlocks = (nhits + blockSize - 1) / blockSize;
    dim3 blks(1, numberOfBlocks, 1);
    dim3 thrs(stride, blockSize, 1);
    gpuPixelDoublets::fishbone<<<blks, thrs, 0, cudaStream>>>(
        hh.view(), device_theCells_.get(), device_nCells_, device_isOuterHitOfCell_.get(), nhits, false);
    cudaCheck(hipGetLastError());
  }

  blockSize = 64;
  numberOfBlocks = (3 * m_params.maxNumberOfDoublets_ / 4 + blockSize - 1) / blockSize;
  kernel_find_ntuplets<<<numberOfBlocks, blockSize, 0, cudaStream>>>(hh.view(),
                                                                     device_theCells_.get(),
                                                                     device_nCells_,
                                                                     device_theCellTracks_,
                                                                     tuples_d,
                                                                     device_hitTuple_apc_,
                                                                     quality_d,
                                                                     m_params.minHitsPerNtuplet_);
  cudaCheck(hipGetLastError());

  if (m_params.doStats_)
    kernel_mark_used<<<numberOfBlocks, blockSize, 0, cudaStream>>>(hh.view(), device_theCells_.get(), device_nCells_);
  cudaCheck(hipGetLastError());

#ifdef GPU_DEBUG
  hipDeviceSynchronize();
  cudaCheck(hipGetLastError());
#endif

  blockSize = 128;
  numberOfBlocks = (HitContainer::totbins() + blockSize - 1) / blockSize;
  cms::cuda::finalizeBulk<<<numberOfBlocks, blockSize, 0, cudaStream>>>(device_hitTuple_apc_, tuples_d);

  // remove duplicates (tracks that share a doublet)
  numberOfBlocks = (3 * m_params.maxNumberOfDoublets_ / 4 + blockSize - 1) / blockSize;
  kernel_earlyDuplicateRemover<<<numberOfBlocks, blockSize, 0, cudaStream>>>(
      device_theCells_.get(), device_nCells_, tuples_d, quality_d);
  cudaCheck(hipGetLastError());

  // fill multiplicity histos 
  static cms::cuda::CoopKernelConfig config(CAConstants::TupleMultiplicity::nthreads());
  auto kc = config.getConfig(kernel_fillMultiplicity);
  blockSize = kc.second;
  numberOfBlocks = int(3 * CAConstants::maxTuples() / 4 + blockSize - 1) / blockSize;
  numberOfBlocks = std::min(numberOfBlocks,kc.first);
  launch_cooperative(kernel_fillMultiplicity,{numberOfBlocks, blockSize, 0, cudaStream},
      tuples_d, quality_d, device_tupleMultiplicity_.get());
  cudaCheck(hipGetLastError());

  if (nhits > 1 && m_params.lateFishbone_) {
    auto nthTot = 128;
    auto stride = 16;
    auto blockSize = nthTot / stride;
    auto numberOfBlocks = (nhits + blockSize - 1) / blockSize;
    dim3 blks(1, numberOfBlocks, 1);
    dim3 thrs(stride, blockSize, 1);
    gpuPixelDoublets::fishbone<<<blks, thrs, 0, cudaStream>>>(
        hh.view(), device_theCells_.get(), device_nCells_, device_isOuterHitOfCell_.get(), nhits, true);
    cudaCheck(hipGetLastError());
  }

  if (m_params.doStats_) {
    numberOfBlocks = (std::max(nhits, m_params.maxNumberOfDoublets_) + blockSize - 1) / blockSize;
    kernel_checkOverflows<<<numberOfBlocks, blockSize, 0, cudaStream>>>(tuples_d,
                                                                        device_tupleMultiplicity_.get(),
                                                                        device_hitTuple_apc_,
                                                                        device_theCells_.get(),
                                                                        device_nCells_,
                                                                        device_theCellNeighbors_,
                                                                        device_theCellTracks_,
                                                                        device_isOuterHitOfCell_.get(),
                                                                        nhits,
                                                                        m_params.maxNumberOfDoublets_,
                                                                        counters_);
    cudaCheck(hipGetLastError());
  }
#ifdef GPU_DEBUG
  hipDeviceSynchronize();
  cudaCheck(hipGetLastError());
#endif
}


template <>
void CAHitNtupletGeneratorKernelsGPU::classifyTuples(HitsOnCPU const &hh, TkSoA *tracks_d, hipStream_t cudaStream) {
  // these are pointer on GPU!
  auto const *tuples_d = &tracks_d->hitIndices;
  auto *quality_d = (Quality *)(&tracks_d->m_quality);

  auto blockSize = 64;

  // classify tracks based on kinematics
  int numberOfBlocks = (3 * CAConstants::maxNumberOfQuadruplets() / 4 + blockSize - 1) / blockSize;
  kernel_classifyTracks<<<numberOfBlocks, blockSize, 0, cudaStream>>>(tuples_d, tracks_d, m_params.cuts_, quality_d);
  cudaCheck(hipGetLastError());

  if (m_params.lateFishbone_) {
    // apply fishbone cleaning to good tracks
    numberOfBlocks = (3 * m_params.maxNumberOfDoublets_ / 4 + blockSize - 1) / blockSize;
    kernel_fishboneCleaner<<<numberOfBlocks, blockSize, 0, cudaStream>>>(
        device_theCells_.get(), device_nCells_, quality_d);
    cudaCheck(hipGetLastError());
  }

  // remove duplicates (tracks that share a doublet)
  numberOfBlocks = (3 * m_params.maxNumberOfDoublets_ / 4 + blockSize - 1) / blockSize;
  kernel_fastDuplicateRemover<<<numberOfBlocks, blockSize, 0, cudaStream>>>(
      device_theCells_.get(), device_nCells_, tuples_d, tracks_d);
  cudaCheck(hipGetLastError());

  if (m_params.minHitsPerNtuplet_ < 4 || m_params.doStats_) {
    // fill hit->track "map"
     static cms::cuda::CoopKernelConfig config(CAConstants::HitToTuple::nthreads());
     auto kc = config.getConfig(kernel_fillHitInTracks);
     blockSize = kc.second;
     numberOfBlocks = int(3 * CAConstants::maxTuples() / 4 + blockSize - 1) / blockSize;
     numberOfBlocks = std::min(numberOfBlocks,kc.first);
     launch_cooperative(kernel_fillHitInTracks,{numberOfBlocks, blockSize, 0, cudaStream},
        tuples_d, quality_d, device_hitToTuple_.get());
     cudaCheck(hipGetLastError());
  }
  if (m_params.minHitsPerNtuplet_ < 4) {
    // remove duplicates (tracks that share a hit)
    numberOfBlocks = (HitToTuple::capacity() + blockSize - 1) / blockSize;
    kernel_tripletCleaner<<<numberOfBlocks, blockSize, 0, cudaStream>>>(
        hh.view(), tuples_d, tracks_d, quality_d, device_hitToTuple_.get());
    cudaCheck(hipGetLastError());
  }

  if (m_params.doStats_) {
    // counters (add flag???)
    numberOfBlocks = (HitToTuple::capacity() + blockSize - 1) / blockSize;
    kernel_doStatsForHitInTracks<<<numberOfBlocks, blockSize, 0, cudaStream>>>(device_hitToTuple_.get(), counters_);
    cudaCheck(hipGetLastError());
    numberOfBlocks = (3 * CAConstants::maxNumberOfQuadruplets() / 4 + blockSize - 1) / blockSize;
    kernel_doStatsForTracks<<<numberOfBlocks, blockSize, 0, cudaStream>>>(tuples_d, quality_d, counters_);
    cudaCheck(hipGetLastError());
  }
#ifdef GPU_DEBUG
  hipDeviceSynchronize();
  cudaCheck(hipGetLastError());
#endif

#ifdef DUMP_GPU_TK_TUPLES
  static std::atomic<int> iev(0);
  ++iev;
  kernel_print_found_ntuplets<<<1, 32, 0, cudaStream>>>(
      hh.view(), tuples_d, tracks_d, quality_d, device_hitToTuple_.get(), 100, iev);
#endif
}

template <>
void CAHitNtupletGeneratorKernelsGPU::printCounters(Counters const *counters) {
  kernel_printCounters<<<1, 1>>>(counters);
}
