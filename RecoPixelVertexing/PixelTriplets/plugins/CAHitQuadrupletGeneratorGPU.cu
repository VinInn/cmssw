#include "hip/hip_runtime.h"
//
// Author: Felice Pantaleo, CERN
//

#include "HeterogeneousCore/CUDAUtilities/interface/cudaCheck.h"
#include "CAHitQuadrupletGeneratorGPU.h"
#include "GPUCACell.h"

__global__ void
kernel_connect(GPUCACell *cells, uint32_t const * nCells,
               GPU::VecArray< unsigned int, 512> *isOuterHitOfCell,
               float ptmin, 
               float region_origin_radius, const float thetaCut,
               const float phiCut, const float hardPtCut,
               unsigned int maxNumberOfDoublets_, unsigned int maxNumberOfHits_) {

  float region_origin_x =0.;
  float region_origin_y =0.;

  auto cellIndex = threadIdx.x + blockIdx.x * blockDim.x;
  if (cellIndex >= (*nCells) ) return;
  auto &thisCell = cells[cellIndex];
  auto innerHitId = thisCell.get_inner_hit_id();
  auto numberOfPossibleNeighbors = isOuterHitOfCell[innerHitId].size();
  for (auto j = 0; j < numberOfPossibleNeighbors; ++j) {
     auto otherCell = isOuterHitOfCell[innerHitId][j];

     if (thisCell.check_alignment_and_tag(
                 cells, otherCell, ptmin, region_origin_x, region_origin_y,
                  region_origin_radius, thetaCut, phiCut, hardPtCut)
        ) {
          cells[otherCell].theOuterNeighbors.push_back(cellIndex);
     }
  }
}

__global__ void kernel_find_ntuplets(
    GPUCACell *cells, uint32_t const * nCells,
    GPU::SimpleVector<Quadruplet> *foundNtuplets,
    unsigned int *rootLayerPairs, unsigned int minHitsPerNtuplet,
    unsigned int maxNumberOfDoublets_)
{

  auto cellIndex = threadIdx.x + blockIdx.x * blockDim.x;
  if (cellIndex >= (*nCells) ) return;
  auto &thisCell = cells[cellIndex];
  if (thisCell.theLayerPairId!=0 && thisCell.theLayerPairId!=3 && thisCell.theLayerPairId!=8) return; // inner layer is 0 FIXME
  GPU::VecArray<unsigned int, 3> stack;
  thisCell.find_ntuplets(cells, foundNtuplets, stack, minHitsPerNtuplet);

  printf("in %d found quadruplets: %d", cellIndex, foundNtuplets->size());
}

template <int maxNumberOfDoublets_>
__global__ void
kernel_print_found_ntuplets(GPU::SimpleVector<Quadruplet> *foundNtuplets) {
  for (int i = 0; i < foundNtuplets->size(); ++i) {
    printf("\nquadruplet %d: %d %d, %d %d, %d %d\n", i,
           (*foundNtuplets)[i].hitId[0],
           (*foundNtuplets)[i].hitId[1],
           (*foundNtuplets)[i].hitId[2],
           (*foundNtuplets)[i].hitId[3],

  }
}

void CAHitQuadrupletGeneratorGPU::deallocateOnGPU()
{
  hipHostFree(h_indices_);
  hipHostFree(h_doublets_);
  hipHostFree(h_x_);
  hipHostFree(h_y_);
  hipHostFree(h_z_);
  hipHostFree(h_rootLayerPairs_);
  for (size_t i = 0; i < h_foundNtupletsVec_.size(); ++i)
  {
    hipHostFree(h_foundNtupletsVec_[i]);
    hipHostFree(h_foundNtupletsData_[i]);
    hipFree(d_foundNtupletsVec_[i]);
    hipFree(d_foundNtupletsData_[i]);
  }
  hipHostFree(tmp_layers_);
  hipHostFree(tmp_layerDoublets_);
  hipHostFree(h_layers_);

  hipFree(d_indices_);
  hipFree(d_doublets_);
  hipFree(d_layers_);
  hipFree(d_x_);
  hipFree(d_y_);
  hipFree(d_z_);
  hipFree(d_rootLayerPairs_);
  hipFree(device_theCells_);
  hipFree(device_isOuterHitOfCell_);
  hipFree(device_nCells_);
}

void CAHitQuadrupletGeneratorGPU::allocateOnGPU()
{
  cudaCheck(hipHostMalloc(&h_doublets_, maxNumberOfLayerPairs_ * sizeof(GPULayerDoublets)));
  cudaCheck(hipHostMalloc(&h_indices_, maxNumberOfLayerPairs_ * maxNumberOfDoublets_ * 2 * sizeof(int)));
  cudaCheck(hipHostMalloc(&h_x_, maxNumberOfLayers_ * maxNumberOfHits_ * sizeof(float)));
  cudaCheck(hipHostMalloc(&h_y_, maxNumberOfLayers_ * maxNumberOfHits_ * sizeof(float)));
  cudaCheck(hipHostMalloc(&h_z_, maxNumberOfLayers_ * maxNumberOfHits_ * sizeof(float)));
  cudaCheck(hipHostMalloc(&h_rootLayerPairs_, maxNumberOfRootLayerPairs_ * sizeof(int)));

  cudaCheck(hipMalloc(&d_indices_, maxNumberOfLayerPairs_ * maxNumberOfDoublets_ * 2 * sizeof(int)));
  cudaCheck(hipMalloc(&d_doublets_, maxNumberOfLayerPairs_ * sizeof(GPULayerDoublets)));
  cudaCheck(hipMalloc(&d_layers_, maxNumberOfLayers_ * sizeof(GPULayerHits)));
  cudaCheck(hipMalloc(&d_x_, maxNumberOfLayers_ * maxNumberOfHits_ * sizeof(float)));
  cudaCheck(hipMalloc(&d_y_, maxNumberOfLayers_ * maxNumberOfHits_ * sizeof(float)));
  cudaCheck(hipMalloc(&d_z_, maxNumberOfLayers_ * maxNumberOfHits_ * sizeof(float)));
  cudaCheck(hipMalloc(&d_rootLayerPairs_, maxNumberOfRootLayerPairs_ * sizeof(unsigned int)));

  //////////////////////////////////////////////////////////
  // ALLOCATIONS FOR THE INTERMEDIATE RESULTS (STAYS ON WORKER)
  //////////////////////////////////////////////////////////

  cudaCheck(hipMalloc(&device_theCells_,
             maxNumberOfLayerPairs_ * maxNumberOfDoublets_ * sizeof(GPUCACell)));
  cudaCheck(hipMalloc(&device_nCells_,sizeof(uint32_t)));

  cudaCheck(hipMalloc(&device_isOuterHitOfCell_,
             maxNumberOfLayers_ * maxNumberOfHits_ * sizeof(GPU::VecArray<unsigned int, maxCellsPerHit_>)));
  cudaCheck(hipMemset(device_isOuterHitOfCell_, 0,
             maxNumberOfLayers_ * maxNumberOfHits_ * sizeof(GPU::VecArray<unsigned int, maxCellsPerHit_>)));

  h_foundNtupletsVec_.resize(maxNumberOfRegions_);
  h_foundNtupletsData_.resize(maxNumberOfRegions_);
  d_foundNtupletsVec_.resize(maxNumberOfRegions_);
  d_foundNtupletsData_.resize(maxNumberOfRegions_);

  // FIXME this could be rewritten with a single pair of hipHostMalloc / hipMalloc
  for (int i = 0; i < maxNumberOfRegions_; ++i) {
    cudaCheck(hipHostMalloc(&h_foundNtupletsData_[i],  sizeof(Quadruplet) * maxNumberOfQuadruplets_));
    cudaCheck(hipHostMalloc(&h_foundNtupletsVec_[i],   sizeof(GPU::SimpleVector<Quadruplet>)));
    new(h_foundNtupletsVec_[i]) GPU::SimpleVector<Quadruplet>(maxNumberOfQuadruplets_, h_foundNtupletsData_[i]);
    cudaCheck(hipMalloc(&d_foundNtupletsData_[i],      sizeof(Quadruplet) * maxNumberOfQuadruplets_));
    cudaCheck(hipMemset(d_foundNtupletsData_[i], 0x00, sizeof(Quadruplet) * maxNumberOfQuadruplets_));
    cudaCheck(hipMalloc(&d_foundNtupletsVec_[i],       sizeof(GPU::SimpleVector<Quadruplet>)));
    GPU::SimpleVector<Quadruplet> tmp_foundNtuplets(maxNumberOfQuadruplets_, d_foundNtupletsData_[i]);
    cudaCheck(hipMemcpy(d_foundNtupletsVec_[i], & tmp_foundNtuplets, sizeof(GPU::SimpleVector<Quadruplet>), hipMemcpyDefault));
  }

  cudaCheck(hipHostMalloc(&tmp_layers_, maxNumberOfLayers_ * sizeof(GPULayerHits)));
  cudaCheck(hipHostMalloc(&tmp_layerDoublets_,maxNumberOfLayerPairs_ * sizeof(GPULayerDoublets)));
  cudaCheck(hipHostMalloc(&h_layers_, maxNumberOfLayers_ * sizeof(GPULayerHits)));
}

void CAHitQuadrupletGeneratorGPU::launchKernels(const TrackingRegion &region,
                                                int regionIndex, hipStream_t cudaStream)
{
  assert(regionIndex < maxNumberOfRegions_);
  dim3 numberOfBlocks_create(64, numberOfLayerPairs_);
//  dim3 numberOfBlocks_connect(32, numberOfLayerPairs_);
  dim3 numberOfBlocks_find(16, numberOfRootLayerPairs_);
  h_foundNtupletsVec_[regionIndex]->reset();
  /*
  kernel_create<<<numberOfBlocks_create, 32, 0, cudaStream>>>(
      numberOfLayerPairs_, d_doublets_, d_layers_, device_theCells_,
      device_isOuterHitOfCell_, d_foundNtupletsVec_[regionIndex],
      region.origin().x(), region.origin().y(), maxNumberOfDoublets_,
      maxNumberOfHits_);
  */

  auto numberOfBlocks = (maxNumberOfDoublets_ + 512 - 1)/512;
  kernel_connect<<<numberOfBlocks, 512, 0, cudaStream>>>(
      device_theCells_, device_nCells_,
      device_isOuterHitOfCell_,
      region.ptMin(), 
      region.originRBound(), caThetaCut, caPhiCut, caHardPtCut,
      maxNumberOfDoublets_, maxNumberOfHits_
  );

  kernel_find_ntuplets<<<numberOfBlocks, 512, 0, cudaStream>>>(
      device_theCells_, device_nCells_,
      d_foundNtupletsVec_[regionIndex],
      d_rootLayerPairs_, 4, maxNumberOfDoublets_);

  cudaCheck(hipMemcpyAsync(h_foundNtupletsVec_[regionIndex], d_foundNtupletsVec_[regionIndex],
                            sizeof(GPU::SimpleVector<Quadruplet>),
                            hipMemcpyDeviceToHost, cudaStream));

  cudaCheck(hipMemcpyAsync(h_foundNtupletsData_[regionIndex], d_foundNtupletsData_[regionIndex],
                            maxNumberOfQuadruplets_*sizeof(Quadruplet),
                            hipMemcpyDeviceToHost, cudaStream));

}

std::vector<std::array<int, 4>>
CAHitQuadrupletGeneratorGPU::fetchKernelResult(int regionIndex, hipStream_t cudaStream)
{
  h_foundNtupletsVec_[regionIndex]->set_data(h_foundNtupletsData_[regionIndex]);
  // this lazily resets temporary memory for the next event, and is not needed for reading the output
  cudaCheck(hipMemsetAsync(device_isOuterHitOfCell_, 0,
                            maxNumberOfLayers_ * maxNumberOfHits_ * sizeof(GPU::VecArray<unsigned int, maxCellsPerHit_>),
                            cudaStream));
  std::vector<std::array<int, 4>> quadsInterface(h_foundNtupletsVec_[regionIndex]->size());
  for (int i = 0; i < h_foundNtupletsVec_[regionIndex]->size(); ++i) {
    for (int j = 0; j<4; ++j) quadsInterface[i][j] = (*h_foundNtupletsVec_[regionIndex])[i].hitId[j];
  }
  return quadsInterface;
}

void CAHitQuadrupletGeneratorGPU::buildDoublets(HitsOnCPU const & hh, hipStream_t stream) {
   auto nhits = hh.nHits;

  int threadsPerBlock = 256;
  int blocks = (3*nhits + threadsPerBlock - 1) / threadsPerBlock;

  cudaCheck(hipMemset(device_nCells_,0,sizeof(uint32_t)));
  gpuPixelDoublets::getDoubletsFromHisto<<<blocks, threadsPerBlock, 0, stream>>>(device_theCells_,device_nCells_,hh.gpu_d, device_isOuterHitOfCell_);
}
