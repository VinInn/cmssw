#include "hip/hip_runtime.h"
//
// Author: Felice Pantaleo, CERN
//

#include <cstdint>
#include <hip/hip_runtime.h>

#include "HeterogeneousCore/CUDAUtilities/interface/cudaCheck.h"
#include "HeterogeneousCore/CUDAUtilities/interface/cuda_assert.h"
#include "RecoLocalTracker/SiPixelRecHits/interface/pixelCPEforGPU.h"
#include "CAHitQuadrupletGeneratorGPU.h"
#include "GPUCACell.h"
#include "gpuPixelDoublets.h"
#include"gpuFishbone.h"
using namespace gpuPixelDoublets;

using HitsOnCPU = siPixelRecHitsHeterogeneousProduct::HitsOnCPU;
using TuplesOnGPU = pixelTuplesHeterogeneousProduct::TuplesOnGPU;
using Quality = pixelTuplesHeterogeneousProduct::Quality;



__global__
void kernel_checkOverflows(TuplesOnGPU::Container * foundNtuplets, AtomicPairCounter * apc,
               GPUCACell const * __restrict__ cells, uint32_t const * __restrict__ nCells,
               GPUCACell::OuterHitOfCell const * __restrict__ isOuterHitOfCell,
               uint32_t nHits, uint32_t maxNumberOfDoublets) {

 __shared__ uint32_t killedCell;
 killedCell=0;
 __syncthreads();
  
 auto idx = threadIdx.x + blockIdx.x * blockDim.x;
 #ifdef GPU_DEBUG
 if (0==idx) {
   printf("number of found cells %d, found tuples %d with total hits %d,%d\n",*nCells, apc->get().m, foundNtuplets->size(), apc->get().n);
   assert(foundNtuplets->size(apc->get().m)==0);
   assert(foundNtuplets->size()==apc->get().n);
 }

 if(idx<foundNtuplets->nbins()) {
   if (foundNtuplets->size(idx)>5) printf("ERROR %d, %d\n", idx, foundNtuplets->size(idx));
   assert(foundNtuplets->size(idx)<6);
   for (auto ih = foundNtuplets->begin(idx); ih!=foundNtuplets->end(idx); ++ih) assert(*ih<nHits);
 }
 #endif


 if (idx < (*nCells) ) {
   auto &thisCell = cells[idx];
   if (thisCell.theOuterNeighbors.full()) //++tooManyNeighbors[thisCell.theLayerPairId];
     printf("OuterNeighbors overflow %d in %d\n", idx, thisCell.theLayerPairId);
   if (thisCell.theTracks.full()) //++tooManyTracks[thisCell.theLayerPairId];
     printf("Tracks overflow %d in %d\n", idx, thisCell.theLayerPairId);
   if (thisCell.theDoubletId<0) atomicInc(&killedCell,maxNumberOfDoublets);
 }
 if (idx < nHits) {
   if (isOuterHitOfCell[idx].full()) // ++tooManyOuterHitOfCell;
     printf("OuterHitOfCell overflow %d\n", idx);
 }

 __syncthreads();
// if (threadIdx.x==0) printf("number of killed cells %d\n",killedCell);
}

__global__
void
kernel_fastDuplicateRemover(GPUCACell const * cells, uint32_t const * __restrict__ nCells,
                            Rfit::helix_fit const * __restrict__ hfit,
                            pixelTuplesHeterogeneousProduct::Quality * quality
                           ) {

   constexpr auto bad = pixelTuplesHeterogeneousProduct::bad;
   constexpr auto dup = pixelTuplesHeterogeneousProduct::dup;
   constexpr auto loose = pixelTuplesHeterogeneousProduct::loose;

  auto cellIndex = threadIdx.x + blockIdx.x * blockDim.x;

  if (cellIndex >= (*nCells) ) return;
  auto const & thisCell = cells[cellIndex];
  if (thisCell.theDoubletId<0) return;

  // find min chi2
  float mc=1000.f; uint16_t im=60000;
  for (auto it : thisCell.theTracks) {
    if (quality[it]!= bad && hfit[it].chi2_line+hfit[it].chi2_circle < mc) {
      mc=hfit[it].chi2_line+hfit[it].chi2_circle;
      im=it;
    }
  }
  // mark duplicates
  for (auto it : thisCell.theTracks) {
     if (it!=im) quality[it] = dup; //no race:  simple assignment of the same constant
  }
}

__global__ 
void
kernel_connect(AtomicPairCounter * apc1, AtomicPairCounter * apc2,  // just to zero them,
               GPUCACell::Hits const *  __restrict__ hhp,
               GPUCACell * cells, uint32_t const * __restrict__ nCells,
               GPUCACell::OuterHitOfCell const * __restrict__ isOuterHitOfCell,
               float ptmin,
               float region_origin_radius, const float thetaCut,
               const float phiCut, const float hardPtCut,
               unsigned int maxNumberOfDoublets_, unsigned int maxNumberOfHits_) {

  auto const & hh = *hhp;

  // 87 cm/GeV = 1/(3.8T * 0.3)
  // take less than radius given by the hardPtCut and reject everything below
  // auto hardCurvCut = 1.f/(hardPtCut * 87.f);
  constexpr auto hardCurvCut = 1.f/(0.35f * 87.f); // VI tune

  auto cellIndex = threadIdx.x + blockIdx.x * blockDim.x;

  if (0==cellIndex) { (*apc1)=0; (*apc2)=0; }// ready for next kernel

  if (cellIndex >= (*nCells) ) return;
  auto const & thisCell = cells[cellIndex];
  if (thisCell.theDoubletId<0) return;
  auto innerHitId = thisCell.get_inner_hit_id();
  auto numberOfPossibleNeighbors = isOuterHitOfCell[innerHitId].size();
  auto vi = isOuterHitOfCell[innerHitId].data();
  for (auto j = 0; j < numberOfPossibleNeighbors; ++j) {
     auto otherCell = __ldg(vi+j);
     if (cells[otherCell].theDoubletId<0) continue;
     if (thisCell.check_alignment(hh,
                 cells[otherCell], ptmin,
                  region_origin_radius+phiCut, thetaCut, hardCurvCut)
        ) {
          cells[otherCell].theOuterNeighbors.push_back(cellIndex);
     }
  }
}

__global__ 
void kernel_find_ntuplets(
    GPUCACell * __restrict__ cells, uint32_t const * nCells,
    TuplesOnGPU::Container * foundNtuplets, AtomicPairCounter * apc,
    unsigned int minHitsPerNtuplet,
    unsigned int maxNumberOfDoublets_)
{

  auto cellIndex = threadIdx.x + blockIdx.x * blockDim.x;
  if (cellIndex >= (*nCells) ) return;
  auto &thisCell = cells[cellIndex];
  if (thisCell.theLayerPairId!=0 && thisCell.theLayerPairId!=3 && thisCell.theLayerPairId!=8) return; // inner layer is 0 FIXME
  GPUCACell::TmpTuple stack;
  stack.reset();
  thisCell.find_ntuplets(cells, *foundNtuplets, *apc, stack, minHitsPerNtuplet);
  assert(stack.size()==0);
  // printf("in %d found quadruplets: %d\n", cellIndex, apc->get());
}


__global__
void kernel_VerifyFit(TuplesOnGPU::Container const * __restrict__ tuples,
                 Rfit::helix_fit const *  __restrict__ fit_results,
                 Quality *  __restrict__ quality) {

  auto idx = blockIdx.x * blockDim.x + threadIdx.x;
  if (idx>= tuples->nbins()) return;
  if (tuples->size(idx)==0) {
    return;
  }

  quality[idx] = pixelTuplesHeterogeneousProduct::bad;

  // only quadruplets
  if (tuples->size(idx)<4) { 
    return;
  }

  bool isNaN = false;
  for (int i=0; i<5; ++i) {
    isNaN |=  fit_results[idx].par(i)!=fit_results[idx].par(i);
  }
  isNaN |=  !(fit_results[idx].chi2_line+fit_results[idx].chi2_circle < 1000.f);  // catch NaN as well
  quality[idx] = isNaN ? quality[idx] : pixelTuplesHeterogeneousProduct::loose;

}

__global__
void kernel_print_found_ntuplets(GPU::SimpleVector<Quadruplet> *foundNtuplets, int maxPrint) {
  for (int i = 0; i < std::min(maxPrint, foundNtuplets->size()); ++i) {
    printf("\nquadruplet %d: %d %d %d %d\n", i,
           (*foundNtuplets)[i].hitId[0],
           (*foundNtuplets)[i].hitId[1],
           (*foundNtuplets)[i].hitId[2],
           (*foundNtuplets)[i].hitId[3]
          );

  }
}

void CAHitQuadrupletGeneratorGPU::deallocateOnGPU()
{

  fitter.deallocateOnGPU();

  hipFree(device_theCells_);
  hipFree(device_isOuterHitOfCell_);
  hipFree(device_nCells_);
  hipFree(device_cellToTuple_);
  hipFree(device_cellToTuple_apc_);

  //product
  hipFree(gpu_.tuples_d);
  hipFree(gpu_.helix_fit_results_d);
  hipFree(gpu_.apc_d);
  hipFree(gpu_d);
  hipFree(tuples_);
  hipFree(helix_fit_results_);
}

void CAHitQuadrupletGeneratorGPU::allocateOnGPU()
{
  //////////////////////////////////////////////////////////
  // ALLOCATIONS FOR THE INTERMEDIATE RESULTS (STAYS ON WORKER)
  //////////////////////////////////////////////////////////

  cudaCheck(hipMalloc(&device_theCells_,
             maxNumberOfLayerPairs_ * maxNumberOfDoublets_ * sizeof(GPUCACell)));
  cudaCheck(hipMalloc(&device_nCells_, sizeof(uint32_t)));
  cudaCheck(hipMemset(device_nCells_, 0, sizeof(uint32_t)));

  cudaCheck(hipMalloc(&device_isOuterHitOfCell_,
             PixelGPUConstants::maxNumberOfHits * sizeof(GPU::VecArray<unsigned int, maxCellsPerHit_>)));
  cudaCheck(hipMemset(device_isOuterHitOfCell_, 0,
             PixelGPUConstants::maxNumberOfHits * sizeof(GPU::VecArray<unsigned int, maxCellsPerHit_>)));

   cudaCheck(hipMalloc(&device_cellToTuple_, sizeof(CellToTuple)));
   cudaCheck(hipMalloc(&device_cellToTuple_apc_, sizeof(AtomicPairCounter)));

  //product
  cudaCheck(hipMalloc(&gpu_.tuples_d, sizeof(TuplesOnGPU::Container)));
  cudaCheck(hipMalloc(&gpu_.apc_d, sizeof(AtomicPairCounter)));
  cudaCheck(hipMalloc(&gpu_.helix_fit_results_d, sizeof(Rfit::helix_fit)*maxNumberOfQuadruplets_));
  cudaCheck(hipMalloc(&gpu_.quality_d, sizeof(Quality)*maxNumberOfQuadruplets_));

  cudaCheck(hipMalloc(&gpu_d, sizeof(TuplesOnGPU)));
  gpu_.me_d = gpu_d;
  cudaCheck(hipMemcpy(gpu_d, &gpu_, sizeof(TuplesOnGPU), hipMemcpyDefault));

  cudaCheck(hipHostMalloc(&tuples_, sizeof(TuplesOnGPU::Container)));
  cudaCheck(hipHostMalloc(&helix_fit_results_, sizeof(Rfit::helix_fit)*maxNumberOfQuadruplets_));
  cudaCheck(hipHostMalloc(&quality_, sizeof(Quality)*maxNumberOfQuadruplets_));

  fitter.allocateOnGPU(gpu_.tuples_d, gpu_.helix_fit_results_d);


}

void CAHitQuadrupletGeneratorGPU::launchKernels(const TrackingRegion &region,
                                                int regionIndex, HitsOnCPU const & hh,
                                                bool doRiemannFit,
                                                bool transferToCPU,
                                                hipStream_t cudaStream)
{
  assert(regionIndex < maxNumberOfRegions_);
  assert(0==regionIndex);


  auto nhits = hh.nHits;
  assert(nhits <= PixelGPUConstants::maxNumberOfHits);
  auto blockSize = 64;
  auto stride = 4;
  auto numberOfBlocks = (nhits + blockSize - 1)/blockSize;
  numberOfBlocks *=stride;
  fishbone<<<numberOfBlocks, blockSize, 0, cudaStream>>>(
      hh.gpu_d,
      device_theCells_, device_nCells_,
      device_isOuterHitOfCell_,
      nhits, stride
  );

  numberOfBlocks = (maxNumberOfDoublets_ + blockSize - 1)/blockSize;
  kernel_connect<<<numberOfBlocks, blockSize, 0, cudaStream>>>(
      gpu_.apc_d, device_cellToTuple_apc_,  // needed only to be reset, ready for next kernel
      hh.gpu_d,
      device_theCells_, device_nCells_,
      device_isOuterHitOfCell_,
      region.ptMin(),
      region.originRBound(), caThetaCut, caPhiCut, caHardPtCut,
      maxNumberOfDoublets_, PixelGPUConstants::maxNumberOfHits
  );
  cudaCheck(hipGetLastError());

  kernel_find_ntuplets<<<numberOfBlocks, blockSize, 0, cudaStream>>>(
      device_theCells_, device_nCells_,
      gpu_.tuples_d,
      gpu_.apc_d,
      4, maxNumberOfDoublets_);
  cudaCheck(hipGetLastError());

  numberOfBlocks = (TuplesOnGPU::Container::totbins() + blockSize - 1)/blockSize;
  cudautils::finalizeBulk<<<numberOfBlocks, blockSize, 0, cudaStream>>>(gpu_.apc_d,gpu_.tuples_d);

  numberOfBlocks = (std::max(int(nhits), maxNumberOfDoublets_) + blockSize - 1)/blockSize;
  kernel_checkOverflows<<<numberOfBlocks, blockSize, 0, cudaStream>>>(
                        gpu_.tuples_d, gpu_.apc_d,
                        device_theCells_, device_nCells_,
                        device_isOuterHitOfCell_, nhits,
                        maxNumberOfDoublets_
                       );
  cudaCheck(hipGetLastError());

  // kernel_print_found_ntuplets<<<1, 1, 0, cudaStream>>>(gpu_.tuples_d, 10);

  if (doRiemannFit) {
    launchFit(hh, nhits, cudaStream);
    numberOfBlocks = (maxNumberOfQuadruplets_ + blockSize - 1)/blockSize;
    kernel_VerifyFit<<<numberOfBlocks, blockSize, 0, cudaStream>>>(gpu_.tuples_d, gpu_.helix_fit_results_d, gpu_.quality_d);

    numberOfBlocks = (maxNumberOfDoublets_ + blockSize - 1)/blockSize;
    kernel_fastDuplicateRemover<<<numberOfBlocks, blockSize, 0, cudaStream>>>(device_theCells_, device_nCells_,gpu_.helix_fit_results_d, gpu_.quality_d);
  }


  if (transferToCPU) {
    cudaCheck(hipMemcpyAsync(tuples_,gpu_.tuples_d,
                              sizeof(TuplesOnGPU::Container),
                              hipMemcpyDeviceToHost, cudaStream));

    cudaCheck(hipMemcpyAsync(helix_fit_results_,gpu_.helix_fit_results_d, 
                              sizeof(Rfit::helix_fit)*maxNumberOfQuadruplets_,
                              hipMemcpyDeviceToHost, cudaStream));

    cudaCheck(hipMemcpyAsync(quality_,gpu_.quality_d,
                              sizeof(Quality)*maxNumberOfQuadruplets_,
                              hipMemcpyDeviceToHost, cudaStream));

  }

}

void CAHitQuadrupletGeneratorGPU::cleanup(hipStream_t cudaStream) {
  // this lazily resets temporary memory for the next event, and is not needed for reading the output
  cudaCheck(hipMemsetAsync(device_isOuterHitOfCell_, 0,
                            PixelGPUConstants::maxNumberOfHits * sizeof(GPU::VecArray<unsigned int, maxCellsPerHit_>),
                            cudaStream));
  cudaCheck(hipMemsetAsync(device_nCells_, 0, sizeof(uint32_t), cudaStream));
}

std::vector<std::array<int, 4>>
CAHitQuadrupletGeneratorGPU::fetchKernelResult(int)
{
  assert(tuples_);
  auto const & tuples = *tuples_;

  uint32_t sizes[7]={0};
  std::vector<int> ntk(10000);
  auto add = [&](uint32_t hi) { if (hi>=ntk.size()) ntk.resize(hi+1); ++ntk[hi];};

  std::vector<std::array<int, 4>> quadsInterface; quadsInterface.reserve(10000);

  nTuples_=0;
  for (auto i = 0U; i < tuples.nbins(); ++i) {
    auto sz = tuples.size(i);
    if (sz==0) break;  // we know cannot be less then 3
    ++nTuples_;
    ++sizes[sz];
    for (auto j=tuples.begin(i); j!=tuples.end(i); ++j) add(*j);
    if (sz<4) continue;
    quadsInterface.emplace_back(std::array<int, 4>());
    quadsInterface.back()[0] = tuples.begin(i)[0];
    quadsInterface.back()[1] = tuples.begin(i)[1];
    quadsInterface.back()[2] = tuples.begin(i)[2];   // [sz-2];
    quadsInterface.back()[3] = tuples.begin(i)[3];   // [sz-1];
  }

//#ifdef GPU_DEBUG
  long long ave =0; int nn=0; for (auto k : ntk) if(k>0){ave+=k; ++nn;}
  std::cout << "Q Produced " << quadsInterface.size() << " quadruplets: ";
  for (auto i=3; i<7; ++i) std::cout << sizes[i] << ' ';
  std::cout << "max/ave " << *std::max_element(ntk.begin(),ntk.end())<<'/'<<float(ave)/float(nn) << std::endl;
//#endif
  return quadsInterface;
}

void CAHitQuadrupletGeneratorGPU::buildDoublets(HitsOnCPU const & hh, hipStream_t stream) {
  auto nhits = hh.nHits;

  int threadsPerBlock = gpuPixelDoublets::getDoubletsFromHistoMaxBlockSize;
  int blocks = (3 * nhits + threadsPerBlock - 1) / threadsPerBlock;
  gpuPixelDoublets::getDoubletsFromHisto<<<blocks, threadsPerBlock, 0, stream>>>(device_theCells_, device_nCells_, hh.gpu_d, device_isOuterHitOfCell_);
  cudaCheck(hipGetLastError());
}
