#include "gpuClusterTracks.h"

namespace gpuVertexFinder {


  void Producer::allocateOnGPU() {
    cudaCheck(hipMalloc(&onGPU.zt, OnGPU::MAXTRACKS*sizeof(float)));
    cudaCheck(hipMalloc(&onGPU.ezt2, OnGPU::MAXTRACKS*sizeof(float)));
    cudaCheck(hipMalloc(&onGPU.iv, OnGPU::MAXTRACKS*sizeof(int32_t)));

    cudaCheck(hipMalloc(&onGPU.nv, sizeof(uint32_t)));
    cudaCheck(hipMalloc(&onGPU.zv, OnGPU::MAXVTX*sizeof(float)));
    cudaCheck(hipMalloc(&onGPU.wv, OnGPU::MAXVTX*sizeof(float)));
 

    cudaCheck(hipMalloc(&onGPU.izt, OnGPU::MAXTRACKS*sizeof(int8_t)));
    cudaCheck(hipMalloc(&onGPU.nn, OnGPU::MAXTRACKS*sizeof(uint16_t)));

    cudaCheck(hipMalloc(&onGPU_d,sizeof(OnGPU)));
    cudaCheck(hipMemcpy(onGPU_d,&onGPU,sizeof(OnGPU),hipMemcpyHostToDevice));

  }
	      
  void Producer::deallocateOnGPU() {
    cudaCheck(hipFree(onGPU.zt));
    cudaCheck(hipFree(onGPU.ezt2));
    cudaCheck(hipFree(onGPU.iv));

    cudaCheck(hipFree(onGPU.nv));
    cudaCheck(hipFree(onGPU.zv));
    cudaCheck(hipFree(onGPU.wv));
 

    cudaCheck(hipFree(onGPU.izt));
    cudaCheck(hipFree(onGPU.nn));

    cudaCheck(hipFree(onGPU));

  }


  void Producer::produce(hipStream_t stream,
			 float const * zt,
			 float const * ezt2,
			 uint32_t ntrks
			 ) {
    

    cudaCheck(hipMemcpyAsync((onGPU.zt,zt,ntrks*sizeof(float),
			       hipMemcpyHostToDevice,stream)));
    cudaCheck(hipMemcpyAsync((onGPU.ezt2,ezt2,ntrks*sizeof(float),
			       hipMemcpyHostToDevice,stream)));

    clusterTracks<<<1,1024,0,stream>>>(ntrks,onGPU_d,3,0.1f);
    
    cudaCheck(hipMemcpy(&gpuProduct.nVertices,onGPU.nv, sizeof(uint32_t),
			 hipMemcpyDeviceToHost,));

    gpuProduct.z.resize(gpuProduct.nVertices);
    cudaCheck(hipMemcpyAsync(gpuProduct.z.data(),onGPU.zv,sizeof(float)*gpuProduct.nVertices,
			       hipMemcpyDeviceToHost, cudaStream));
    gpuProduct.zerr.resize(gpuProduct.nVertices);
    cudaCheck(hipMemcpyAsync(gpuProduct.zerr.data(),onGPU.wv,sizeof(float)*gpuProduct.nVertices,
			       hipMemcpyDeviceToHost, cudaStream));
    
  }
  
  
} // end namespace

