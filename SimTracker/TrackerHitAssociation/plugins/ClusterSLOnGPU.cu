#include "hip/hip_runtime.h"
#include "ClusterSLOnGPU.h"

// for the "packing"
#include "RecoLocalTracker/SiPixelClusterizer/plugins/SiPixelRawToClusterGPUKernel.h"
#include "HeterogeneousCore/CUDAUtilities/interface/cudastdAlgorithm.h"
#include<cassert>
#include<atomic>

/*
struct ClusterSLGPU {
 ClusterSLGPU(){alloc();}
 void alloc();

 ClusterSLGPU * me_d;
 std::array<uint32_t,4> * links_d;
 uint32_t * tkId_d;
 uint32_t * tkId2_d;
 uint32_t * n1_d;
 uint32_t * n2_d;

 static constexpr uint32_t MAX_DIGIS = 2000*150;
 static constexpr uint32_t MaxNumModules = 2000;

};
*/


template<class ForwardIt, class T, class Compare>
__device__
ForwardIt lowerBound(ForwardIt first, ForwardIt last, const T& value, Compare comp)
{
    ForwardIt it;
    auto count = last-first;
 
    while (count > 0) {
        it = first;
        auto step = count / 2;
        it+=step;
        if (comp(*it, value)) {
            first = ++it;
            count -= step + 1;
        }
        else
            count = step;
    }
    return first;
}



__global__
void simLink(clusterSLOnGPU::DigisOnGPU const * ddp, uint32_t ndigis, clusterSLOnGPU::HitsOnGPU const * hhp, ClusterSLGPU const * slp, uint32_t n) {

  constexpr uint16_t InvId=9999; // must be > MaxNumModules
  
  auto const & dd = *ddp;
  auto const & hh = *hhp;
  auto const & sl = *slp;
  auto i = blockIdx.x*blockDim.x + threadIdx.x;
  
  if (i>ndigis) return;

  auto id = dd.moduleInd_d[i];
  if (InvId==id) return;
  assert(id<2000);

  auto ch = pixelgpudetails::pixelToChannel(dd.xx_d[i], dd.yy_d[i]);
  auto first = hh.hitsModuleStart_d[id];
  auto cl = first + dd.clus_d[i];
  assert(cl<256*2000);
  
  const std::array<uint32_t,4> me{{id,ch,0}};

  auto less = [](std::array<uint32_t,4> const & a, std::array<uint32_t,4> const & b)->bool {
     return a[0]<b[0] || ( !(b[0]<a[0]) && a[1]<b[1]); // in this context we do not care of [2] 
  };

  auto equal = [](std::array<uint32_t,4> const & a, std::array<uint32_t,4> const & b)->bool {
     return a[0]==b[0] && a[1]==b[1]; // in this context we do not care of [2]
  };

  auto const * b = sl.links_d;
  auto const * e = b+n;

  // auto p = cuda_std::lower_bound(b,e,me,less);
  auto p = lowerBound(b,e,me,less);
  auto j = p-sl.links_d;
  assert(j>=0);
  j = std::min(int(j),int(n-1));
  if (equal(me,sl.links_d[j])) {
    //auto const & l = sl.links_d[j];
    auto const tk = j; // l[2];
    auto old = atomicCAS(&sl.tkId_d[cl],0,tk);
    if (0==old ||tk==old) atomicAdd(&sl.n1_d[cl],1);
    else {
      auto old = atomicCAS(&sl.tkId2_d[cl],0,tk);
      if (0==old ||tk==old) atomicAdd(&sl.n2_d[cl],1);
    }    
  } 
  /*
  else {
    auto const & k=sl.links_d[j];
    auto const & kk = j+1<n ? sl.links_d[j+1] : k;
    printf("digi not found %d:%d closest %d:%d:%d, %d:%d:%d\n",id,ch, k[0],k[1],k[2], kk[0],kk[1],kk[2]);
  }
  */

}


__global__
void dumpLink(int ev, clusterSLOnGPU::HitsOnGPU const * hhp, uint32_t nhits, ClusterSLGPU const * slp) {
  auto i = blockIdx.x*blockDim.x + threadIdx.x;
  if (i>nhits) return;

  auto const & hh = *hhp;
  auto const & sl = *slp;

  auto const & tk1 = sl.links_d[sl.tkId_d[i]];
  auto const & tk2 = sl.links_d[sl.tkId2_d[i]];

  printf("HIT: %d %d %d %d %f %f %f %f %d %d %d %d %d %d %d\n",ev, i, 
         hh.detInd_d[i], hh.charge_d[i], 
         hh.xg_d[i],hh.yg_d[i],hh.zg_d[i],hh.rg_d[i],hh.iphi_d[i], 
         tk1[2],tk1[3],sl.n1_d[i],
         tk2[2],tk2[3],sl.n2_d[i]
        );

}



namespace clusterSLOnGPU {

  struct CSVHeader {
     CSVHeader() {
      printf("HIT: %s %s %s %s %s %s %s %s %s %s %s %s %s %s %s\n", "ev", "ind",
         "det", "charge",	
         "xg","yg","zg","rg","iphi", 
         "tkId","pt","n1","tkId2","pt2","n2" 
        );
     }

  };
  CSVHeader csvHeader;

  std::atomic<int> evId(0);

  void wrapper(DigisOnGPU const & dd, uint32_t ndigis, HitsOnCPU const & hh, uint32_t nhits, ClusterSLGPU const & sl, uint32_t n, cuda::stream_t<>& stream) {
    
    int ev = ++evId;
    int threadsPerBlock = 256;
    int blocks = (ndigis + threadsPerBlock - 1) / threadsPerBlock;

    assert(sl.me_d);
    simLink<<<blocks, threadsPerBlock, 0, stream.id()>>>(dd.me_d,ndigis, hh.gpu_d, sl.me_d,n);
    blocks = (nhits + threadsPerBlock - 1) / threadsPerBlock;
    dumpLink<<<blocks, threadsPerBlock, 0, stream.id()>>>(ev, hh.gpu_d, nhits, sl.me_d);
    cudaCheck(hipGetLastError());

  }

}
